#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include "g4c.h"
#include "g4c_mm.hh"

static const char *G4C_SEMCPY = "GPU memory copy failed";
static const char *G4C_SEKERNEL = "GPU kernel launch or execution failed";
static const char *G4C_SENOTEXIST = "no such error";

// G4C configurations:
struct g4c_context {
	int nr_streams;
	hipStream_t *streams;
	int *stream_uses;

	int hostmem_handle;
	size_t hostmem_sz;
	void *hostmem_start;

	int devmem_handle;
	size_t devmem_sz;
	void *devmem_start;
} __cur_ctx;

extern "C" const char *g4c_strerror(int err) {
	switch(err) {
	case G4C_EMCPY:
		return G4C_SEMCPY;
	case G4C_EKERNEL:
		return G4C_SEKERNEL;
	default:
		return G4C_SENOTEXIST;
	}
}

#define csc(...) _cuda_safe_call(__VA_ARGS__, __FILE__, __LINE__)
static hipError_t
_cuda_safe_call(hipError_t e, const char *file, int line) {
    if (e!=hipSuccess) {
	fprintf(stderr, "g4c Error: %s %d %s\n",
		file, line, hipGetErrorString(e));
	hipDeviceReset();
	abort();
    }
    return e;
}


extern "C" int
g4c_init(int nr_ss, size_t hm_sz, size_t dm_sz) {
	int i;

	// Enable memory map, and spin CPU thread when waiting for sync to
	// decrease latency.
	csc( hipSetDeviceFlags(
		     hipDeviceScheduleSpin|hipDeviceMapHost ) );

	// Set up stream management:
	__cur_ctx.nr_streams = nr_ss;
	__cur_ctx.streams = malloc(sizeof(hipStream_t)
				   *(__cur_ctx.nr_streams+1));
	__cur_ctx.stream_uses = malloc(sizeof(int)
				       *(__cur_ctx.nr_streams+1));
	if (!__cur_ctx.streams || !__cur_ctx.stream_uses)
		return -ENOMEM;
	
	// Create streams
	for (i=1; i<__cur_ctx.nr_streams+1; i++) {
		csc( hipStreamCreate(&__cur_ctx.streams[i]) );
		__cur_ctx.stream_uses[i] = 0;
	}

	// Set up MM:
	__cur_ctx.hostmem_sz = hm_sz;
	__cur_ctx.devmem_sz = dm_sz;

	csc( hipHostAlloc(&__cur_ctx.hostmem_start,
			   hm_sz, hipHostMallocPortable) );
	csc( hipMalloc(&__cur_ctx.devmem_start,
			dm_sz) );

	__cur_ctx.hostmem_handle =
		g4c_new_mm_handle(
			__cur_ctx.hostmem_start,
			hm_sz,
			G4C_PAGE_SHIFT);
	__cur_ctx.devmem_handle =
		g4c_new_mm_handle(
			__cur_ctx.devmem_start,
			dm_sz,
			G4C_PAGE_SHIFT);

	csc( hipGetLastError() );

	return 0;
}

extern "C" void
g4c_exit(void) {
	int i;

	for (i=1; i<__cur_ctx.nr_streams+1; i++) {
		csc( hipStreamDestroy(__cur_ctx.streams[i]) );
	}

	g4c_release_mm_handle(__cur_ctx.hostmem_handle);
	g4c_release_mm_handle(__cur_ctx.devmem_handle);

	csc( hipFree(__cur_ctx.devmem_start) );
	csc( hipHostFree(__cur_ctx.hostmem_start) );
}

extern "C" int
g4c_stream_done(int s) {
	hipError_t e = hipStreamQuery(__cur_ctx.streams[s]);
	if (e == hipSuccess) {
		return 1;
	} else if (e != hipErrorNotReady) {
		csc(e);
	}

	return 0;
}

extern "C" int
g4c_stream_sync(int s) {
	csc( hipStreamSynchronize(__cur_ctx.streams[s]) );
	return 0;
}

extern "C" int
g4c_h2d_async(void *h, void *d, size_t sz, int s)
{
	csc( hipMemcpyAsync(d, h, sz, hipMemcpyHostToDevice,
			     __cur_ctx.streams[s]) );
	return 0;
}

extern "C" int
g4c_d2h_async(void *d, void *h, size_t sz, int s)
{
	csc( hipMemcpyAsync(h, d, sz, hipMemcpyDeviceToHost,
			     __cur_ctx.streams[s]) );
	return 0;
}

extern "C" int
g4c_dev_memset(void *d, int val, size_t sz, int s)
{
	csc( hipMemsetAsync(d, val, sz, __cur_ctx.streams[s]) );
	return 0;
}

extern "C" int
g4c_alloc_stream()
{
	for (int i=1; i<__cur_ctx.nr_streams+1; i++) {
		if (__cur_ctx.stream_uses[i] == 0) {
			__cur_ctx.stream_uses[i] = 1;
			return i;
		}
	}

	return 0;
}

extern "C" void
g4c_free_stream(int s)
{
	__cur_ctx.stream_uses[s] = 0;
}



// Memory management functions.
extern "C" void *
g4c_alloc_page_lock_mem(size_t sz)
{
	return g4c_alloc_mem(__cur_ctx.hostmem_handle, sz);
}

extern "C" void
g4c_free_page_lock_mem(void *p)
{
	g4c_free_mem(__cur_ctx.hostmem_handle, p);
}

extern "C" void *
g4c_alloc_dev_mem(size_t sz)
{
	return g4c_alloc_mem(__cur_ctx.devmem_handle, sz);
}

extern "C" void
g4c_free_dev_mem(void *p)
{
	g4c_free_mem(__cur_ctx.devmem_handle, p);
}

// End of file.
