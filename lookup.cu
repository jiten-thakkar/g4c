#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "lookup.h"
#include "g4c.hh"

template<int BITS>
class trie_node {
public:
    int id;
    uint8_t port;
    int bits;
    trie_node<BITS> *children[(1<<BITS)];
    trie_node<BITS> *prev;
    trie_node<BITS> *next;
    
    trie_node():bits(BITS), prev(0), next(0) {
	for (int i=0; i<(1<<BITS); i++)
	    children[i] = 0;	
    }
};

template<int BITS>
class node_store {
public:
    int id_seq;
    int bits;
    trie_node<BITS> head;
    node_store() : id_seq(0),
		   bits(BITS) {
	head.prev = &head;
	head.next = &head;
    }

    void append(trie_node<BITS> *n) {
	n->next = &head;
	n->prev = head.prev;
	head.prev->next = n;
	head.prev = n;
    }

    template<class node_type> g4c_lpm_tree *
    build_lpm_tree(g4c_ipv4_rt_entry *ents, int n, uint8_t fport,
		   node_type dummy) {
	trie_node<BITS> *root = new trie_node<BITS>();
	root->id = id_seq++;
	root->port = fport;
	append(root);

	trie_node<BITS> *node = root;
	for (int i=0; i<n; i++) {
	    uint32_t val = 0;
	    int ite;
	    for_bits_h2l(32-BITS, 32-ents[i].nnetbits,
			 ents[i].addr, val, ite, BITS) {
		if (!node->children[val]) {
		    trie_node<BITS> *p = new trie_node<BITS>();
		    p->id = id_seq++;
		    p->port = fport;
		    append(p);
		    node->children[val] = p;
		}
		node = node->children[val];
	    }
	    node->port = ents[i].port;
	}   

	g4c_lpm_tree * lpmt = (g4c_lpm_tree*)malloc(
	    sizeof(g4c_lpm_tree) + id_seq*sizeof(node_type));
	if (!lpmt)
	    return 0;

	lpmt->nbits = BITS;
	lpmt->nnodes = id_seq;
	lpmt->fport = fport;
	node = root;
	node_type *tnds = (node_type*)lpmt->nodes.b1;
	for (int i=0; i<lpmt->nnodes; i++) {
	    if (node == 0) { // report problem
		free(lpmt);
		return 0;
	    }
	
	    tnds[i].port = node->port;
	    for (int j=0; j<(1<<BITS); j++) {
		if (node->children[j])
		    tnds[i].children[j] = node->children[j]->id;
		else
		    tnds[i].children[j] = 0;
	    }
	    node = node->next;	
	}

	return lpmt;
    }

    void clear() {
	trie_node<BITS> *node = head.next;
	while (node != &head) {
	    node = node->next;
	    delete node->prev;
	}
	head.next = head.prev = 0;
    }    
};

template<class node_type, int BITS> static g4c_lpm_tree *
__build_lpm_tree(node_store<BITS> store,
		 g4c_ipv4_rt_entry *ents, int n, uint8_t fport,
		 node_type dummy)
{
    g4c_lpm_tree * lpmt = store.build_lpm_tree(ents, n, fport, dummy);
    store.clear();
    return lpmt;
}

extern "C" g4c_lpm_tree *
g4c_build_lpm_tree(g4c_ipv4_rt_entry *ents, int n, int nbits, uint8_t fport)
{    
    switch(nbits) {
    case 1:
	return __build_lpm_tree(
	    node_store<1>(), ents, n, fport, g4c_lpm_1b_node());
    case 2:
	return __build_lpm_tree(
	    node_store<2>(), ents, n, fport, g4c_lpm_2b_node());
    case 4:
	return __build_lpm_tree(
	    node_store<4>(), ents, n, fport, g4c_lpm_4b_node());
    default:
	return 0;
    }
}

template<class node_type> uint8_t
__ipv4_lookup(g4c_lpm_tree *lpmtrie, uint32_t addr, node_type dummy)
{
    uint32_t val;
    int ite;

    node_type *node = (node_type*)lpmtrie->nodes.b1;
    int nid = 0;
    uint8_t port = lpmtrie->fport;
    for_bits_h2l(32-lpmtrie->nbits, 0, addr, val, ite, lpmtrie->nbits) {
	if (node[nid].port != lpmtrie->fport)
	    port = node[nid].port;
	
	if (node[nid].children[val]) {
	    nid = node[nid].children[val];
	} else
	    break;
    }

    return port;
}

extern "C" uint8_t
g4c_ipv4_lookup(g4c_lpm_tree *lpmtrie, uint32_t addr)
{
    switch(lpmtrie->nbits) {
    case 1:
	return __ipv4_lookup(lpmtrie, addr, g4c_lpm_1b_node());
    case 2:
	return __ipv4_lookup(lpmtrie, addr, g4c_lpm_2b_node());
    case 4:
	return __ipv4_lookup(lpmtrie, addr, g4c_lpm_4b_node());
    default:
	return lpmtrie->fport;
    }
}


template <typename node_type> __global__ void
gpu_lpm_lookup(g4c_lpm_tree *lpmt,
	       uint32_t *addrs,
	       uint8_t *ports,
	       int n, node_type *dummy)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    uint32_t val, addr = addrs[id];
    int ite, nid=0;
    node_type *node = (node_type*)lpmt->nodes.b1;
    for_bits_h2l(32-lpmt->nbits, 0, addr, val, ite, lpmt->nbits) {
	if (node[nid].children[val]) {
	    nid = node[nid].children[val];
	} else {
	    ports[id] = node[nid].port;
	    break;
	}
    }
}


extern "C" int
g4c_ipv4_gpu_lookup(g4c_lpm_tree *dlpmt,
		    uint32_t *daddrs,
		    uint8_t *dports,
		    int nbits, int n, int s)
{
    hipStream_t stream = g4c_get_stream(s);

    switch(nbits) {
    case 1:
	gpu_lpm_lookup<<<n/32, 32, 0, stream>>>(
	    dlpmt, daddrs, dports, n, (g4c_lpm_1b_node*)(0));
	break;
    case 2:
	gpu_lpm_lookup<<<n/32, 32, 0, stream>>>(
	    dlpmt, daddrs, dports, n, (g4c_lpm_2b_node*)(0));
	break;
    case 4:
	gpu_lpm_lookup<<<n/32, 32, 0, stream>>>(
	    dlpmt, daddrs, dports, n, (g4c_lpm_4b_node*)(0));
	break;
    default:
	gpu_lpm_lookup<<<n/32, 32, 0, stream>>>(
	    dlpmt, daddrs, dports, n, (g4c_lpm_1b_node*)(0));
	break;
    }

    return 0;
}


// Import as soon as we need
#include <vector>
#include <algorithm>
using namespace std;

static bool
__ipv4_rt_ent_less(g4c_ipv4_rt_entry *e1, g4c_ipv4_rt_entry *e2)
{
    return e1->addr < e2->addr;
}

extern "C" int
g4c_build_static_routing_table(g4c_ipv4_rt_entry *ents, int n,
			       uint8_t fport, uint32_t *srt)
{
    vector<g4c_ipv4_rt_entry *> vents;
    g4c_ipv4_rt_entry first, last;

    first.addr = 0;
    first.port = fport;
    last.addr = (uint32_t)(~0);
    last.port = fport;
    
    vents.reserve(n+2);
    vents.push_back(&first);
    
    for (int i=0; i<n; i++)
	vents.push_back(ents+i);
    sort(vents.begin()+1, vents.end(), __ipv4_rt_ent_less);
    vents.push_back(&last);

    vector<g4c_ipv4_rt_entry *>::iterator n1 = vents.begin();
    vector<g4c_ipv4_rt_entry *>::iterator n2 = n1+1;
    do {
	for (int i = ((*n1)->addr>>8);
	     i < ((*n2)->addr>>8); i++)
	    srt[i] = g4c_srt_entry2((*n1)->addr, (*n1)->port);
	++n1;
	++n2;	
    } while (n1 != vents.end());

    return 0;
}

extern "C" uint8_t
g4c_ipv4_static_lookup(uint32_t *srt, uint32_t addr)
{
    return g4c_srt_port(srt[g4c_srt_subnet_idx(addr)]);
}

__global__ void
gpu_static_lookup(uint32_t *srt, uint32_t *addrs, uint8_t *ports, int n)
{
}

extern "C" int
g4c_ipv4_gpu_static_lookup(uint32_t *dsrt, uint32_t *daddrs,
			   uint8_t *dports, int n, int s)
{
    hipStream_t stream = g4c_get_stream(s);
    gpu_static_lookup<<<n/32, 32, 0, stream>>>(dsrt, daddrs, dports, n);
    return 0;
}
